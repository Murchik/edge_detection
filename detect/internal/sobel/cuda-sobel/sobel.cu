#include "hip/hip_runtime.h"
#include <stdint.h>

#include <cstdio>
#include <cstring>

#include "cuda_kernels.h"
#include "sobel.h"

#define CHECK_CUDART_ERROR(call)                                       \
    do {                                                               \
        hipError_t status = call;                                     \
        if (status != hipSuccess) {                                   \
            fprintf(stderr, "ERROR in %s:%d. Message: %s\n", __FILE__, \
                    __LINE__, hipGetErrorString(status));             \
            return 1;                                                  \
        }                                                              \
    } while (0)

int ApplySobel(uint32_t *data, int w, int h) {
    // Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar4>();
    hipArray_t cuArray;
    CHECK_CUDART_ERROR(hipMallocArray(&cuArray, &channelDesc, w, h));

    // Set pitch of the source 
    // (the width in memory in bytes of the 2D array pointed to by src, including padding)
    const size_t spitch = w * sizeof(uchar4);
    // Copy data located in host memory to device memory
    CHECK_CUDART_ERROR(hipMemcpy2DToArray(cuArray, 0, 0, data, spitch, w * sizeof(uchar4), h, hipMemcpyHostToDevice));

    // Specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType         = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    // Specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeWrap;
    texDesc.addressMode[1]   = hipAddressModeWrap;
    texDesc.filterMode       = hipFilterModePoint;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    // Create texture object
    hipTextureObject_t texObj = 0;
    CHECK_CUDART_ERROR(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL));

    // Allocate result of transformation in device memory
    uchar4 *output;
    CHECK_CUDART_ERROR(hipMalloc(&output, w * h * sizeof(uchar4)));

    // Invoke kernel
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((w + threadsperBlock.x - 1) / threadsperBlock.x,
                   (h + threadsperBlock.y - 1) / threadsperBlock.y);
    greyscaleKernel<<<numBlocks, threadsperBlock>>>(output, texObj, w, h);

    // Copy data from device back to host
    CHECK_CUDART_ERROR(hipMemcpy(data, output, w * h * sizeof(uint32_t), hipMemcpyDeviceToHost));

    // Destroy texture object
    CHECK_CUDART_ERROR(hipDestroyTextureObject(texObj));

    // Free device memory
    CHECK_CUDART_ERROR(hipFreeArray(cuArray));
    CHECK_CUDART_ERROR(hipFree(output));

    return 0;
}
