#include "hip/hip_runtime.h"
#include <stdint.h>

#include <cstdio>
#include <cstring>

#include "kernels.h"
#include "sobel.h"

#define CHECK_CUDART_ERROR(call)                                       \
    do {                                                               \
        hipError_t status = call;                                     \
        if (status != hipSuccess) {                                   \
            fprintf(stderr, "ERROR in %s:%d. Message: %s\n", __FILE__, \
                    __LINE__, hipGetErrorString(status));             \
            return 1;                                                  \
        }                                                              \
    } while (0)

int createTextureObject(hipTextureObject_t& TexObj, const hipArray_t& cuArray) {
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType         = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeClamp;
    texDesc.addressMode[1]   = hipAddressModeClamp;
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    CHECK_CUDART_ERROR(hipCreateTextureObject(&TexObj, &resDesc, &texDesc, NULL));

    return 0;
}

int createSurfaceObject(hipSurfaceObject_t& SurfObj, hipArray_t& cuArray) {
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType         = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    CHECK_CUDART_ERROR(hipCreateSurfaceObject(&SurfObj, &resDesc));

    return 0;
}

template <typename T>
int createCudaArray(hipArray_t& cuArray, int width, int height) {
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc<T>();
    CHECK_CUDART_ERROR(hipMallocArray(&cuArray, &channel_desc, width, height));
    return 0;
}

int applyToFloat(const hipArray_t& inCuArray, hipArray_t& outCuArray, int width, int height) {
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width  + threadsperBlock.x - 1) / threadsperBlock.x,
                   (height + threadsperBlock.y - 1) / threadsperBlock.y);

    hipTextureObject_t inTexObj = 0;
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType         = hipResourceTypeArray;
    resDesc.res.array.array = inCuArray;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeClamp;
    texDesc.addressMode[1]   = hipAddressModeClamp;
    texDesc.filterMode       = hipFilterModePoint;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    CHECK_CUDART_ERROR(hipCreateTextureObject(&inTexObj, &resDesc, &texDesc, NULL));

    hipSurfaceObject_t outSurfObj = 0;
    createSurfaceObject(outSurfObj, outCuArray);

    toFloatKernel<<<numBlocks, threadsperBlock>>>(inTexObj, outSurfObj, width, height);
    CHECK_CUDART_ERROR(hipDeviceSynchronize());

    CHECK_CUDART_ERROR(hipDestroySurfaceObject(outSurfObj));
    CHECK_CUDART_ERROR(hipDestroyTextureObject(inTexObj));

    return 0;
}

int applyGreyscale(const hipArray_t& inCuArray, hipArray_t& outCuArray, int width, int height) {
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width  + threadsperBlock.x - 1) / threadsperBlock.x,
                   (height + threadsperBlock.y - 1) / threadsperBlock.y);

    hipTextureObject_t inTexObj = 0;
    createTextureObject(inTexObj, inCuArray);

    hipSurfaceObject_t outSurfObj = 0;
    createSurfaceObject(outSurfObj, outCuArray);

    greyscaleKernel<<<numBlocks, threadsperBlock>>>(inTexObj, outSurfObj, width, height);
    CHECK_CUDART_ERROR(hipDeviceSynchronize());

    CHECK_CUDART_ERROR(hipDestroySurfaceObject(outSurfObj));
    CHECK_CUDART_ERROR(hipDestroyTextureObject(inTexObj));

    return 0;
}

int applyGaussian(const hipArray_t& inCuArray, hipArray_t& outCuArray, int width, int height) {
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width + threadsperBlock.x - 1) / threadsperBlock.x,
                   (height + threadsperBlock.y - 1) / threadsperBlock.y);

    hipTextureObject_t inTexObj = 0;
    createTextureObject(inTexObj, inCuArray);

    hipSurfaceObject_t outSurfObj = 0;
    createSurfaceObject(outSurfObj, outCuArray);

    gaussianKernel<<<numBlocks, threadsperBlock>>>(inTexObj, outSurfObj, width, height);
    CHECK_CUDART_ERROR(hipDeviceSynchronize());

    CHECK_CUDART_ERROR(hipDestroySurfaceObject(outSurfObj));
    CHECK_CUDART_ERROR(hipDestroyTextureObject(inTexObj));

    return 0;
}

// int applySobel(const hipArray_t& inCuArray, hipArray_t& outCuArray, int width, int height) {
//     dim3 threadsperBlock(16, 16);
//     dim3 numBlocks((width + threadsperBlock.x - 1) / threadsperBlock.x,
//                    (height + threadsperBlock.y - 1) / threadsperBlock.y);

//     // Applying vertical kernel of the Sobel filter
//     hipTextureObject_t inTexObj = 0;
//     createTextureObject(inTexObj, inCuArray);    

//     hipArray_t outSobelHorizontal;
//     createCudaArray<float>(outSobelHorizontal, nullptr, width, height);
//     hipSurfaceObject_t outSobelHorizontalSurfObj;
//     createSurfaceObject(outSobelHorizontalSurfObj, outSobelHorizontal);

//     sobelHorizontalKernel<<<numBlocks, threadsperBlock>>>(inTexObj, outSobelHorizontalSurfObj, width, height);
//     CHECK_CUDART_ERROR(hipDeviceSynchronize());

//     CHECK_CUDART_ERROR(hipDestroySurfaceObject(outSobelHorizontalSurfObj));
//     CHECK_CUDART_ERROR(hipDestroyTextureObject(inTexObj));

//     // Applying horizontal kernel of the Sobel filter
//     hipArray_t inCuArrayCopy;
//     createCudaArray<float>(inCuArrayCopy, inCuArray, width, height);
//     hipTextureObject_t inTexObjCopy = 0;
//     createTextureObject(inTexObjCopy, inCuArrayCopy);

//     hipArray_t outSobelVertical;
//     createCudaArray<float>(outSobelVertical, nullptr, width, height);
//     hipSurfaceObject_t outSobelVerticalSurfObj;
//     createSurfaceObject(outSobelVerticalSurfObj, outSobelVertical);

//     sobelVerticalKernel<<<numBlocks, threadsperBlock>>>(inTexObjCopy, outSobelVerticalSurfObj, width, height);
//     CHECK_CUDART_ERROR(hipDeviceSynchronize());

//     CHECK_CUDART_ERROR(hipDestroySurfaceObject(outSobelVerticalSurfObj));
//     CHECK_CUDART_ERROR(hipDestroyTextureObject(inTexObjCopy));

//     // Getting result of the Sobel filter
//     hipSurfaceObject_t outSurfObj;
//     createSurfaceObject(outSurfObj, outCuArray);

//     hipTextureObject_t inHorizontalTexObj = 0;
//     createTextureObject(inHorizontalTexObj, outSobelHorizontal);

//     hipTextureObject_t inVerticalTexObj = 0;
//     createTextureObject(inVerticalTexObj, outSobelVertical);

//     rootKernel<<<numBlocks, threadsperBlock>>>(inHorizontalTexObj, inVerticalTexObj, outSurfObj, width, height);
//     CHECK_CUDART_ERROR(hipDeviceSynchronize());

//     CHECK_CUDART_ERROR(hipDestroyTextureObject(inVerticalTexObj));
//     CHECK_CUDART_ERROR(hipDestroyTextureObject(inHorizontalTexObj));
//     CHECK_CUDART_ERROR(hipDestroySurfaceObject(outSurfObj));

//     CHECK_CUDART_ERROR(hipFreeArray(outSobelVertical));
//     CHECK_CUDART_ERROR(hipFreeArray(inCuArrayCopy));
//     CHECK_CUDART_ERROR(hipFreeArray(outSobelHorizontal));
// }

int convertToRGBA(const hipArray_t& inCuArray, hipArray_t& outCuArray, int width, int height) {
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width + threadsperBlock.x - 1) / threadsperBlock.x,
                   (height + threadsperBlock.y - 1) / threadsperBlock.y);

    hipTextureObject_t inTexObj = 0;
    createTextureObject(inTexObj, inCuArray);

    hipSurfaceObject_t outSurfObj = 0;
    createSurfaceObject(outSurfObj, outCuArray);

    toRGBaKernel<<<numBlocks, threadsperBlock>>>(inTexObj, outSurfObj, width, height);
    CHECK_CUDART_ERROR(hipDeviceSynchronize());

    CHECK_CUDART_ERROR(hipDestroySurfaceObject(outSurfObj));
    CHECK_CUDART_ERROR(hipDestroyTextureObject(inTexObj));
    
    return 0;
}

int ApplySobel(uint32_t* data, int w, int h) {
    // Copy data to device
    hipArray_t gpuData;
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc<uchar4>();
    CHECK_CUDART_ERROR(hipMallocArray(&gpuData, &channel_desc, w, h));

    const size_t spitch = w * sizeof(uint32_t);
    CHECK_CUDART_ERROR(hipMemcpy2DToArray(gpuData, 0, 0, data, spitch, w * sizeof(uchar4), h, hipMemcpyHostToDevice));
        
    hipArray_t gpuDataFloat4;
    createCudaArray<float4>(gpuDataFloat4, w, h);
    applyToFloat(gpuData, gpuDataFloat4, w, h);
    CHECK_CUDART_ERROR(hipFreeArray(gpuData));

    // Applying greyscale filter
    hipArray_t greyscaleOut;
    createCudaArray<float>(greyscaleOut, w, h);
    applyGreyscale(gpuDataFloat4, greyscaleOut, w, h);
    CHECK_CUDART_ERROR(hipFreeArray(gpuDataFloat4));

    // Applying gaussian blur filter
    hipArray_t gaussianOut;
    createCudaArray<float>(gaussianOut, w, h);
    applyGaussian(greyscaleOut, gaussianOut, w, h);
    CHECK_CUDART_ERROR(hipFreeArray(greyscaleOut));

    // // Applying sobel filter
    // hipArray_t sobelOut;
    // createCudaArray<float>(sobelOut, w, h);
    // applySobel(gaussianOut, sobelOut, w, h);
    // CHECK_CUDART_ERROR(hipFreeArray(gaussianOut));

    // Convert back to RGBA
    hipArray_t output;
    createCudaArray<uchar4>(output, w, h);
    convertToRGBA(gaussianOut, output, w, h);
    CHECK_CUDART_ERROR(hipFreeArray(gaussianOut));

    // Copy data back to host
    CHECK_CUDART_ERROR(hipMemcpy2DFromArray(data, w * sizeof(uint32_t), output, 0, 0, w * sizeof(uchar4), h, hipMemcpyDeviceToHost));
    CHECK_CUDART_ERROR(hipFreeArray(output));

    return 0;
}
