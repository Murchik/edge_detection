#include "hip/hip_runtime.h"
#include <stdint.h>

#include <cmath>
#include <cstdio>
#include <cstring>

#include "hip/hip_runtime.h"
#include "sobel.h"

#define DEBUG 0

#define CHECK_CUDART_ERROR(call)                                       \
    do {                                                               \
        hipError_t status = call;                                     \
        if (status != hipSuccess) {                                   \
            fprintf(stderr, "ERROR in %s:%d. Message: %s\n", __FILE__, \
                    __LINE__, hipGetErrorString(status));             \
            return 1;                                                  \
        }                                                              \
    } while (0)

__global__ void monochrome_kernel(uchar4 *input, float *output, int image_size,
                                  int w) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < image_size) {
        output[i] = 0.299 * (float)input[i].x + 0.587 * (float)input[i].y +
                    0.114 * (float)input[i].z;
    }
}

__global__ void gaussian_kernel(const float *input, float *output, int image_size, int w, int h) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int i = idx / w;
    int j = idx % w;
    if (i < image_size && i > 1 && j > 1 && i < h - 1 && j < w - 1) {
        *(output + j + w * i) = (*(input + (j - 1) + w * (i - 1)) * 1 +
                                *(input + (j    ) + w * (i - 1)) * 2 +
                                *(input + (j + 1) + w * (i - 1)) * 1 +

                                *(input + (j - 1) + w * (i    )) * 2 +
                                *(input + (j    ) + w * (i    )) * 4 +
                                *(input + (j + 1) + w * (i    )) * 2 +

                                *(input + (j - 1) + w * (i + 1)) * 1 +
                                *(input + (j    ) + w * (i + 1)) * 2 +
                                *(input + (j + 1) + w * (i + 1)) * 1) / 16.0;
    }
}

__global__ void sobel_vertical_kernel(const float *input, float *output,
                                      int image_size, int w, int h) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int i = idx / w;
    int j = idx % w;
    if (idx < image_size && i > 1 && j > 1 && i < h - 1 && j < w - 1) {
        *(output + j + w * i) = *(input + (j - 1) + w * (i - 1)) *  1 +
                                *(input + (j    ) + w * (i - 1)) *  0 +
                                *(input + (j + 1) + w * (i - 1)) * -1 +

                                *(input + (j - 1) + w * (i    )) *  2 +
                                *(input + (j    ) + w * (i    )) *  0 +
                                *(input + (j + 1) + w * (i    )) * -2 +

                                *(input + (j - 1) + w * (i + 1)) *  1 +
                                *(input + (j    ) + w * (i + 1)) *  0 +
                                *(input + (j + 1) + w * (i + 1)) * -1;
    }
}

__global__ void sobel_horizontal_kernel(const float *input, float *output,
                                        int image_size, int w, int h) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int i = idx / w;
    int j = idx % w;
    if (idx < image_size && i > 1 && j > 1 && i < h - 1 && j < w - 1) {
        *(output + j + w * i) = *(input + (j - 1) + w * (i - 1)) *  1 +
                                *(input + (j    ) + w * (i - 1)) *  2 +
                                *(input + (j + 1) + w * (i - 1)) *  1 +

                                *(input + (j - 1) + w * (i    )) *  0 +
                                *(input + (j    ) + w * (i    )) *  0 +
                                *(input + (j + 1) + w * (i    )) *  0 +

                                *(input + (j - 1) + w * (i + 1)) * -1 +
                                *(input + (j    ) + w * (i + 1)) * -2 +
                                *(input + (j + 1) + w * (i + 1)) * -1;
    }
}

__global__ void root_kernel(const float *input_vertical,
                            const float *input_horizontal, float *output,
                            int image_size, int w) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int i = idx / w;
    int j = idx % w;
    if (idx < image_size) {
        *(output + j + w * i) = sqrtf(powf(*(input_vertical + j + w * i), 2.0) +
                        powf(*(input_horizontal + j + w * i), 2.0));
    }
}

__global__ void conv_float_uchar4(const float *input, uchar4 *output,
                                  int image_size, int w) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int i = idx / w;
    int j = idx % w;
    if (idx < image_size) {
        float Y = *(input + j + w * i);
        *(output + j + w * i) = make_uchar4(Y, Y, Y, 0);
    }
}

int ApplySobel(uint32_t *data, int w, int h) {
    hipStream_t stream;
    CHECK_CUDART_ERROR(hipStreamCreate(&stream));

    int image_size = w * h;

    size_t image_byte_size = sizeof(uchar4) * image_size;
    int image_float_size = sizeof(float) * image_size;

    int threadsPerBlock = 256;
    int numBlocks = image_size / threadsPerBlock + 1;

    // Copying input data into GPU
    uchar4 *gpu_data;
    CHECK_CUDART_ERROR(hipMalloc(&gpu_data, image_byte_size));
    CHECK_CUDART_ERROR(hipMemcpyAsync(gpu_data, data, image_byte_size,
                                       hipMemcpyHostToDevice, stream));

    CHECK_CUDART_ERROR(hipStreamSynchronize(stream));

    // Сolor (RGBA) to grayscale (float value) conversion
    float *output_monochrome;
    CHECK_CUDART_ERROR(hipMalloc(&output_monochrome, image_float_size));

    monochrome_kernel<<<numBlocks, threadsPerBlock, 0, stream>>>(
        gpu_data, output_monochrome, image_size, w);

    CHECK_CUDART_ERROR(hipStreamSynchronize(stream));

    // Gaussian blur
    float *output_gaussian;
    CHECK_CUDART_ERROR(hipMalloc(&output_gaussian, image_float_size));

    gaussian_kernel<<<numBlocks, threadsPerBlock, 0, stream>>>(
        output_monochrome, output_gaussian, image_size, w, h);

    CHECK_CUDART_ERROR(hipStreamSynchronize(stream));

    // Sobel vertical core
    float *output_vertical;
    CHECK_CUDART_ERROR(hipMalloc(&output_vertical, image_float_size));

    sobel_vertical_kernel<<<numBlocks, threadsPerBlock, 0, stream>>>(
        output_monochrome, output_vertical, image_size, w, h);

    CHECK_CUDART_ERROR(hipStreamSynchronize(stream));

    // Sobel horizontal core
    float *output_horizontal;
    CHECK_CUDART_ERROR(hipMalloc(&output_horizontal, image_float_size));

    sobel_horizontal_kernel<<<numBlocks, threadsPerBlock, 0, stream>>>(
        output_monochrome, output_horizontal, image_size, w, h);

    CHECK_CUDART_ERROR(hipStreamSynchronize(stream));

    // Composing vertical and horizontal outputs in an image
    float *output_sobel;
    CHECK_CUDART_ERROR(hipMalloc(&output_sobel, image_float_size));

    root_kernel<<<numBlocks, threadsPerBlock, 0, stream>>>(
        output_vertical, output_horizontal, output_sobel, image_size, w);

    CHECK_CUDART_ERROR(hipStreamSynchronize(stream));

    // Convert float data back to RGBA
    uchar4 *output;
    CHECK_CUDART_ERROR(hipMalloc(&output, image_byte_size));

    conv_float_uchar4<<<numBlocks, threadsPerBlock, 0, stream>>>(
        output_sobel, output, image_size, w);

    CHECK_CUDART_ERROR(hipStreamSynchronize(stream));

    // Copying output data into CPU
    CHECK_CUDART_ERROR(hipMemcpyAsync(data, output, image_byte_size,
                                       hipMemcpyDeviceToHost, stream));

    CHECK_CUDART_ERROR(hipStreamSynchronize(stream));

    CHECK_CUDART_ERROR(hipStreamDestroy(stream));

    CHECK_CUDART_ERROR(hipFree(gpu_data));
    CHECK_CUDART_ERROR(hipFree(output_monochrome));
    CHECK_CUDART_ERROR(hipFree(output_gaussian));
    CHECK_CUDART_ERROR(hipFree(output_vertical));
    CHECK_CUDART_ERROR(hipFree(output_horizontal));
    CHECK_CUDART_ERROR(hipFree(output_sobel));
    CHECK_CUDART_ERROR(hipFree(output));

    return 0;
}
