#include "hip/hip_runtime.h"
#include <stdint.h>

#include <cstdio>
#include <cstring>

#include "kernels.h"
#include "sobel.h"

#define CHECK_CUDART_ERROR(call)                                       \
    do {                                                               \
        hipError_t status = call;                                     \
        if (status != hipSuccess) {                                   \
            fprintf(stderr, "ERROR in %s:%d. Message: %s\n", __FILE__, \
                    __LINE__, hipGetErrorString(status));             \
            return 1;                                                  \
        }                                                              \
    } while (0)

int createTextureObject(hipTextureObject_t& TexObj, hipArray_t& cuArray)
{
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType         = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeWrap;
    texDesc.addressMode[1]   = hipAddressModeWrap;
    texDesc.filterMode       = hipFilterModePoint;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    CHECK_CUDART_ERROR(hipCreateTextureObject(&TexObj, &resDesc, &texDesc, NULL));

    return 0;
}

int createSurfaceObject(hipSurfaceObject_t& SurfObj, hipArray_t& cuArray)
{
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;

    resDesc.res.array.array = cuArray;
    CHECK_CUDART_ERROR(hipCreateSurfaceObject(&SurfObj, &resDesc));

    return 0;
}

template <typename T>
int initializeCudaArray(hipArray_t& cuArray, const void* data, int width, int height) {
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc<T>();
    CHECK_CUDART_ERROR(hipMallocArray(&cuArray, &channel_desc, width, height));

    if (data) {
        const size_t spitch = width * sizeof(T);
        CHECK_CUDART_ERROR(hipMemcpy2DToArray(cuArray, 0, 0, data, spitch, width * sizeof(T), height, hipMemcpyHostToDevice));
    }

    return 0;
}

int ApplySobel(uint32_t *data, int w, int h) {
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((w + threadsperBlock.x - 1) / threadsperBlock.x,
                   (h + threadsperBlock.y - 1) / threadsperBlock.y);

    hipArray_t gpuData;
    initializeCudaArray<uchar4>(gpuData, data, w, h);

    hipTextureObject_t gpuDataTexObj = 0;
    createTextureObject(gpuDataTexObj, gpuData);

    hipArray_t output;
    initializeCudaArray<uchar4>(output, nullptr, w, h);

    hipSurfaceObject_t outSurfObj = 0;
    createSurfaceObject(outSurfObj, output);

    greyscaleKernel<<<numBlocks, threadsperBlock>>>(gpuDataTexObj, outSurfObj, w, h);
    CHECK_CUDART_ERROR(hipDeviceSynchronize());

    CHECK_CUDART_ERROR(hipMemcpy2DFromArray(data, w * sizeof(uint32_t), output, 0, 0, w * sizeof(uchar4), h, hipMemcpyDeviceToHost));

    CHECK_CUDART_ERROR(hipDestroySurfaceObject(outSurfObj));

    CHECK_CUDART_ERROR(hipFreeArray(output));
    CHECK_CUDART_ERROR(hipFreeArray(gpuData));

    return 0; 
}
