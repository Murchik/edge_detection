#include "hip/hip_runtime.h"
#include <stdint.h>

#include <cstdio>
#include <cstring>

#include "sobel.h"
#include "utils.cu"

int ApplySobel(uint32_t* data, int w, int h) {
    // Copy data to device
    hipArray_t gpuData;
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc<uchar4>();
    CHECK_CUDART_ERROR(hipMallocArray(&gpuData, &channel_desc, w, h));

    const size_t spitch = w * sizeof(uint32_t);
    CHECK_CUDART_ERROR(hipMemcpy2DToArray(gpuData, 0, 0, data, spitch, w * sizeof(uchar4), h, hipMemcpyHostToDevice));
    
    // Convert data to float values
    hipArray_t gpuDataFloat4;
    createCudaArray<float4>(gpuDataFloat4, nullptr, w, h);
    convertToFloat(gpuData, gpuDataFloat4, w, h);
    CHECK_CUDART_ERROR(hipFreeArray(gpuData));

    // Applying greyscale filter
    hipArray_t greyscaleOut;
    createCudaArray<float>(greyscaleOut, nullptr, w, h);
    applyKernel(greyscaleKernel, gpuDataFloat4, greyscaleOut, w, h);
    CHECK_CUDART_ERROR(hipFreeArray(gpuDataFloat4));

    // Applying gaussian blur filter
    hipArray_t gaussianOut;
    createCudaArray<float>(gaussianOut, nullptr, w, h);
    applyKernel(gaussianKernel, greyscaleOut, gaussianOut, w, h);
    CHECK_CUDART_ERROR(hipFreeArray(greyscaleOut));

    // Applying edge detecting filter
    hipArray_t edgeOut;
    createCudaArray<float2>(edgeOut, nullptr, w, h);
    applyKernel(sobelKernel, gaussianOut, edgeOut, w, h);
    CHECK_CUDART_ERROR(hipFreeArray(gaussianOut));

    // Convert back to RGBA
    hipArray_t output;
    createCudaArray<uchar4>(output, nullptr, w, h);
    applyKernel(convertToRGBaKernel, edgeOut, output, w, h);
    CHECK_CUDART_ERROR(hipFreeArray(edgeOut));

    // Copy data back to host
    CHECK_CUDART_ERROR(hipMemcpy2DFromArray(data, w * sizeof(uint32_t), output, 0, 0, w * sizeof(uchar4), h, hipMemcpyDeviceToHost));
    CHECK_CUDART_ERROR(hipFreeArray(output));

    return 0;
}
