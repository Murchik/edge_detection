#include "hip/hip_runtime.h"
#include <stdint.h>

#include <cstdio>
#include <cstring>

#include "kernels.h"
#include "sobel.h"

#define CHECK_CUDART_ERROR(call)                                       \
    do {                                                               \
        hipError_t status = call;                                     \
        if (status != hipSuccess) {                                   \
            fprintf(stderr, "ERROR in %s:%d. Message: %s\n", __FILE__, \
                    __LINE__, hipGetErrorString(status));             \
            return 1;                                                  \
        }                                                              \
    } while (0)

int createTextureObject(hipTextureObject_t& TexObj, const hipArray_t& cuArray) {
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType         = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeWrap;
    texDesc.addressMode[1]   = hipAddressModeWrap;
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    CHECK_CUDART_ERROR(hipCreateTextureObject(&TexObj, &resDesc, &texDesc, NULL));

    return 0;
}

int createSurfaceObject(hipSurfaceObject_t& SurfObj, hipArray_t& cuArray) {
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType         = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    CHECK_CUDART_ERROR(hipCreateSurfaceObject(&SurfObj, &resDesc));

    return 0;
}

template <typename T>
int createCudaArray(hipArray_t& cuArray, const void* data, int width, int height) {
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc<T>();
    CHECK_CUDART_ERROR(hipMallocArray(&cuArray, &channel_desc, width, height));
    if (data) {
        const size_t spitch = width * sizeof(T);
        CHECK_CUDART_ERROR(hipMemcpy2DToArray(cuArray, 0, 0, data, spitch, width * sizeof(T), height, hipMemcpyHostToDevice));
    }

    return 0;
}

int applyToFloat(const hipArray_t& inCuArray, hipArray_t& outCuArray, int width, int height) {
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width  + threadsperBlock.x - 1) / threadsperBlock.x,
                   (height + threadsperBlock.y - 1) / threadsperBlock.y);

    hipTextureObject_t inTexObj = 0;
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType         = hipResourceTypeArray;
    resDesc.res.array.array = inCuArray;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeClamp;
    texDesc.addressMode[1]   = hipAddressModeClamp;
    texDesc.filterMode       = hipFilterModePoint;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    CHECK_CUDART_ERROR(hipCreateTextureObject(&inTexObj, &resDesc, &texDesc, NULL));

    hipSurfaceObject_t outSurfObj = 0;
    createSurfaceObject(outSurfObj, outCuArray);

    toFloatKernel<<<numBlocks, threadsperBlock>>>(inTexObj, outSurfObj, width, height);
    CHECK_CUDART_ERROR(hipDeviceSynchronize());

    CHECK_CUDART_ERROR(hipDestroySurfaceObject(outSurfObj));
    CHECK_CUDART_ERROR(hipDestroyTextureObject(inTexObj));

    return 0;
}

int applyGreyscale(const hipArray_t& inCuArray, hipArray_t& outCuArray, int width, int height) {
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width  + threadsperBlock.x - 1) / threadsperBlock.x,
                   (height + threadsperBlock.y - 1) / threadsperBlock.y);

    hipTextureObject_t inTexObj = 0;
    createTextureObject(inTexObj, inCuArray);

    hipSurfaceObject_t outSurfObj = 0;
    createSurfaceObject(outSurfObj, outCuArray);

    greyscaleKernel<<<numBlocks, threadsperBlock>>>(inTexObj, outSurfObj, width, height);
    CHECK_CUDART_ERROR(hipDeviceSynchronize());

    CHECK_CUDART_ERROR(hipDestroySurfaceObject(outSurfObj));
    CHECK_CUDART_ERROR(hipDestroyTextureObject(inTexObj));

    return 0;
}

int applyGaussian(const hipArray_t& inCuArray, hipArray_t& outCuArray, int width, int height) {
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width + threadsperBlock.x - 1) / threadsperBlock.x,
                   (height + threadsperBlock.y - 1) / threadsperBlock.y);

    hipTextureObject_t inTexObj = 0;
    createTextureObject(inTexObj, inCuArray);

    hipSurfaceObject_t outSurfObj = 0;
    createSurfaceObject(outSurfObj, outCuArray);

    gaussianKernel<<<numBlocks, threadsperBlock>>>(inTexObj, outSurfObj, width, height);
    CHECK_CUDART_ERROR(hipDeviceSynchronize());

    CHECK_CUDART_ERROR(hipDestroySurfaceObject(outSurfObj));
    CHECK_CUDART_ERROR(hipDestroyTextureObject(inTexObj));

    return 0;
}

int applySobelFilter(const hipArray_t& inCuArray, hipArray_t& outCuArray, int width, int height) {
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width + threadsperBlock.x - 1) / threadsperBlock.x,
                   (height + threadsperBlock.y - 1) / threadsperBlock.y);

    hipTextureObject_t inTexObj = 0;
    createTextureObject(inTexObj, inCuArray);

    hipSurfaceObject_t outSurfObj = 0;
    createSurfaceObject(outSurfObj, outCuArray);

    sobelKernel<<<numBlocks, threadsperBlock>>>(inTexObj, outSurfObj, width, height);
    CHECK_CUDART_ERROR(hipDeviceSynchronize());

    CHECK_CUDART_ERROR(hipDestroySurfaceObject(outSurfObj));
    CHECK_CUDART_ERROR(hipDestroyTextureObject(inTexObj));

    return 0;
}

int convertToRGBA(const hipArray_t& inCuArray, hipArray_t& outCuArray, int width, int height) {
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width + threadsperBlock.x - 1) / threadsperBlock.x,
                   (height + threadsperBlock.y - 1) / threadsperBlock.y);

    hipTextureObject_t inTexObj = 0;
    createTextureObject(inTexObj, inCuArray);

    hipSurfaceObject_t outSurfObj = 0;
    createSurfaceObject(outSurfObj, outCuArray);

    toRGBaKernel<<<numBlocks, threadsperBlock>>>(inTexObj, outSurfObj, width, height);
    CHECK_CUDART_ERROR(hipDeviceSynchronize());

    CHECK_CUDART_ERROR(hipDestroySurfaceObject(outSurfObj));
    CHECK_CUDART_ERROR(hipDestroyTextureObject(inTexObj));
    
    return 0;
}

int ApplySobel(uint32_t* data, int w, int h) {
    // Copy data to device
    hipArray_t gpuData;
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc<uchar4>();
    CHECK_CUDART_ERROR(hipMallocArray(&gpuData, &channel_desc, w, h));

    const size_t spitch = w * sizeof(uint32_t);
    CHECK_CUDART_ERROR(hipMemcpy2DToArray(gpuData, 0, 0, data, spitch, w * sizeof(uchar4), h, hipMemcpyHostToDevice));
    
    // Convert data to float values
    hipArray_t gpuDataFloat4;
    createCudaArray<float4>(gpuDataFloat4, nullptr, w, h);
    applyToFloat(gpuData, gpuDataFloat4, w, h);
    CHECK_CUDART_ERROR(hipFreeArray(gpuData));

    // Applying greyscale filter
    hipArray_t greyscaleOut;
    createCudaArray<float>(greyscaleOut, nullptr, w, h);
    applyGreyscale(gpuDataFloat4, greyscaleOut, w, h);
    CHECK_CUDART_ERROR(hipFreeArray(gpuDataFloat4));

    // Applying gaussian blur filter
    hipArray_t gaussianOut;
    createCudaArray<float>(gaussianOut, nullptr, w, h);
    applyGaussian(greyscaleOut, gaussianOut, w, h);
    CHECK_CUDART_ERROR(hipFreeArray(greyscaleOut));

    // Applying sobel filter
    hipArray_t sobelOut;
    createCudaArray<float2>(sobelOut, nullptr, w, h);
    applySobelFilter(gaussianOut, sobelOut, w, h);
    CHECK_CUDART_ERROR(hipFreeArray(gaussianOut));

    // Convert back to RGBA
    hipArray_t output;
    createCudaArray<uchar4>(output, nullptr, w, h);
    convertToRGBA(sobelOut, output, w, h);
    CHECK_CUDART_ERROR(hipFreeArray(sobelOut));

    // Copy data back to host
    CHECK_CUDART_ERROR(hipMemcpy2DFromArray(data, w * sizeof(uint32_t), output, 0, 0, w * sizeof(uchar4), h, hipMemcpyDeviceToHost));
    CHECK_CUDART_ERROR(hipFreeArray(output));

    return 0;
}
