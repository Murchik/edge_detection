#include "hip/hip_runtime.h"
#include <stdint.h>

#include <cstdio>

#include "hip/hip_runtime.h"
#include "sobel.h"

#define CHECK_CUDART_ERROR(call)                                       \
    do {                                                               \
        hipError_t status = call;                                     \
        if (status != hipSuccess) {                                   \
            fprintf(stderr, "ERROR in %s:%d. Message: %s\n", __FILE__, \
                    __LINE__, hipGetErrorString(status));             \
            return 1;                                                  \
        }                                                              \
    } while (0)

/* The index of a thread and its thread ID relate to each other in a
 * straightforward way: For a one-dimensional block, they are the same;
 * for a two-dimensional block of size (Dx, Dy),
 * the thread ID of a thread of index (x, y) is (x + y Dx); */

__global__ void sobel_kernel(uchar4 *data, int w, int h) {
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = tidx; i < w * h; i += stride) {
        float Y = 0.299 * (float)data[i].x + 0.587 * (float)data[i].y +
                  0.114 * (float)data[i].z;
        data[i] = make_uchar4(Y, Y, Y, 0);
    }
}

int ApplySobel(uint32_t *data, int w, int h) {
    hipStream_t stream;
    CHECK_CUDART_ERROR(hipStreamCreate(&stream));
    uchar4 *gpu_data;
    size_t image_byte_size = sizeof(uchar4) * w * h;
    CHECK_CUDART_ERROR(hipMalloc(&gpu_data, image_byte_size));
    CHECK_CUDART_ERROR(hipMemcpyAsync(gpu_data, data, image_byte_size,
                                       hipMemcpyHostToDevice, stream));
    sobel_kernel<<<256, 256, 0, stream>>>(gpu_data, w, h);
    CHECK_CUDART_ERROR(hipMemcpyAsync(data, gpu_data, image_byte_size,
                                       hipMemcpyDeviceToHost, stream));
    CHECK_CUDART_ERROR(hipStreamSynchronize(stream));
    CHECK_CUDART_ERROR(hipStreamDestroy(stream));
    CHECK_CUDART_ERROR(hipFree(gpu_data));
    return 0;
}
