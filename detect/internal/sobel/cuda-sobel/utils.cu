#include "hip/hip_runtime.h"
#ifndef UTILS_CU
#define UTILS_CU

#include <iostream>

#include "timer.hpp"
#include "kernels.cu"

#define CHECK_CUDART_ERROR(call)                                       \
    do {                                                               \
        hipError_t status = call;                                     \
        if (status != hipSuccess) {                                   \
            fprintf(stderr, "ERROR in %s:%d. Message: %s\n", __FILE__, \
                    __LINE__, hipGetErrorString(status));             \
            return 1;                                                  \
        }                                                              \
    } while (0)

int createTextureObject(hipTextureObject_t& TexObj, const hipArray_t& cuArray) {
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType         = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeClamp;
    texDesc.addressMode[1]   = hipAddressModeClamp;
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    CHECK_CUDART_ERROR(hipCreateTextureObject(&TexObj, &resDesc, &texDesc, NULL));

    return 0;
}

int createSurfaceObject(hipSurfaceObject_t& SurfObj, hipArray_t& cuArray) {
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType         = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    CHECK_CUDART_ERROR(hipCreateSurfaceObject(&SurfObj, &resDesc));

    return 0;
}

template <typename T>
int createCudaArray(hipArray_t& cuArray, const void* data, int width, int height) {
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc<T>();
    CHECK_CUDART_ERROR(hipMallocArray(&cuArray, &channel_desc, width, height));
    if (data) {
        const size_t spitch = width * sizeof(T);
        CHECK_CUDART_ERROR(hipMemcpy2DToArray(cuArray, 0, 0, data, spitch, width * sizeof(T), height, hipMemcpyHostToDevice));
    }

    return 0;
}

/*  
    A function that applies the selected kernel to the input hipArray_t array and writes the result to the output hipArray_t
    Please make sure the input and output array types are the same as the kernel input and output arrays. 
        @param K kernel function.
        @param inCuArray hipArray_t of input data.
        @param outCuArray hipArray_t of output data.
        @param width width of the image.
        @param height height of the image.
*/
template <typename K>
int applyKernel(K kernel, const hipArray_t& inCuArray, hipArray_t& outCuArray, int width, int height) {
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks(( width + threadsperBlock.x - 1) / threadsperBlock.x,
                   (height + threadsperBlock.y - 1) / threadsperBlock.y);

    hipTextureObject_t inTexObj = 0;
    createTextureObject(inTexObj, inCuArray);

    hipSurfaceObject_t outSurfObj = 0;
    createSurfaceObject(outSurfObj, outCuArray);

    std::cout << "Invoking a kernel ..." << std::endl;
    {
        timer t;
        kernel<<<numBlocks, threadsperBlock>>>(inTexObj, outSurfObj, width, height);
        CHECK_CUDART_ERROR(hipDeviceSynchronize());
    }

    CHECK_CUDART_ERROR(hipDestroySurfaceObject(outSurfObj));
    CHECK_CUDART_ERROR(hipDestroyTextureObject(inTexObj));

    return 0;
}

/*  
    Conversion from uchar4 to float4 format. A separate function is needed because hipFilterModeLinear cannot be applied to an array of non floats
        @param inCuArray hipArray_t of uchar4.
        @param outCuArray hipArray_t of float4.
*/
int convertToFloat(const hipArray_t& inCuArray, hipArray_t& outCuArray, int width, int height) {
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width  + threadsperBlock.x - 1) / threadsperBlock.x,
                   (height + threadsperBlock.y - 1) / threadsperBlock.y);

    hipTextureObject_t inTexObj = 0;
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType         = hipResourceTypeArray;
    resDesc.res.array.array = inCuArray;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeClamp;
    texDesc.addressMode[1]   = hipAddressModeClamp;
    texDesc.filterMode       = hipFilterModePoint;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    CHECK_CUDART_ERROR(hipCreateTextureObject(&inTexObj, &resDesc, &texDesc, NULL));

    hipSurfaceObject_t outSurfObj = 0;
    createSurfaceObject(outSurfObj, outCuArray);

    convertToFloatKernel<<<numBlocks, threadsperBlock>>>(inTexObj, outSurfObj, width, height);
    CHECK_CUDART_ERROR(hipDeviceSynchronize());

    CHECK_CUDART_ERROR(hipDestroySurfaceObject(outSurfObj));
    CHECK_CUDART_ERROR(hipDestroyTextureObject(inTexObj));

    return 0;
}

#endif // UTILS_CU
